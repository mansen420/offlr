#include <hip/hip_runtime.h>
#include <iostream>

__global__ void foo()
{

}

int main()
{
    foo<<<256, 256>>>();
    auto err = hipGetLastError();
    auto str = hipGetErrorString(err);
    std::cout << str << std::endl;
    return 0;
}
